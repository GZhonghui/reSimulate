#include "hip/hip_runtime.h"
﻿#include"Backend.cuh"

__global__ void UpdateAccelerationKernel(UniversityCUDA_API_Planet* Src, UniversityCUDA_API_Planet* Dst, int Size, float Delta)
{
    int ID = threadIdx.x + blockDim.x * blockIdx.x;

    if(ID < Size)
    {
        Dst[ID].Acceleration.x = 0.0f;
        Dst[ID].Acceleration.y = 0.0f;
        Dst[ID].Acceleration.z = 0.0f;

        int otherIdx = 0;

        UniversityCUDA_API_Planet Me = Src[ID];

        while(otherIdx < Size)
        {
            // if(otherIdx != ID)
            {
                float dX = Src[otherIdx].Location.x - Me.Location.x;
                float dY = Src[otherIdx].Location.y - Me.Location.y;
                float dZ = Src[otherIdx].Location.z - Me.Location.z;

                float distanceSquare = dX * dX + dY * dY + dZ * dZ;

                float Mom = std::powf(distanceSquare + 0.0001, 1.5);
                float Left = Src[otherIdx].Mass / Mom;

                Dst[otherIdx].Acceleration.x += Left * dX;
                Dst[otherIdx].Acceleration.y += Left * dY;
                Dst[otherIdx].Acceleration.z += Left * dZ;
            }
            otherIdx += 1;
        }
    }
}

void UpdateAcceleration(UniversityCUDA_API_Planet* Src, UniversityCUDA_API_Planet* Dst, int Size, float Delta)
{
    UpdateAccelerationKernel<<<20, 1024>>>(Src,Dst,Size,Delta);
}

__global__ void UpdateLocationKernel(UniversityCUDA_API_Planet* Src, UniversityCUDA_API_Planet* Dst, int Size, float Delta)
{
    int ID = threadIdx.x + blockDim.x * blockIdx.x;

    if(ID < Size)
    {
        Dst[ID].Location.x = Src[ID].Location.x + Src[ID].Speed.x * Delta;
        Dst[ID].Location.y = Src[ID].Location.y + Src[ID].Speed.y * Delta;
        Dst[ID].Location.z = Src[ID].Location.z + Src[ID].Speed.z * Delta;
    }
}

void UpdateLocation(UniversityCUDA_API_Planet* Src, UniversityCUDA_API_Planet* Dst, int Size, float Delta)
{
    UpdateLocationKernel<<<20, 1024>>>(Src,Dst,Size,Delta);
}

__global__ void UpdateSpeedKernel(UniversityCUDA_API_Planet* Src, UniversityCUDA_API_Planet* Dst, int Size, float Delta)
{
    int ID = threadIdx.x + blockDim.x * blockIdx.x;

    if(ID < Size)
    {
        Dst[ID].Speed.x = Src[ID].Speed.x + Src[ID].Acceleration.x * Delta;
        Dst[ID].Speed.y = Src[ID].Speed.y + Src[ID].Acceleration.y * Delta;
        Dst[ID].Speed.z = Src[ID].Speed.z + Src[ID].Acceleration.z * Delta;
    }
}

void UpdateSpeed(UniversityCUDA_API_Planet* Src, UniversityCUDA_API_Planet* Dst, int Size, float Delta)
{
    UpdateSpeedKernel<<<20, 1024>>>(Src,Dst,Size,Delta);
}
